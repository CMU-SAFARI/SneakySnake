#include "hip/hip_runtime.h"
/*
 * Copyright (c) <2017 - 2030>, ETH Zurich and Bilkent University
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * Redistributions of source code must retain the above copyright notice, this list
 * of conditions and the following disclaimer.
 * - Redistributions in binary form must reproduce the above copyright notice, this
 *   list of conditions and the following disclaimer in the documentation and/or other
 *   materials provided with the distribution.
 * - Neither the names of the ETH Zurich, Bilkent University,
 *   nor the names of its contributors may be
 *   used to endorse or promote products derived from this software without specific
 *   prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 * LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  Authors: 
  Mohammed Alser
	  mohammed.alser AT inf DOT ethz DOT ch
  Date:
  September 22nd, 2019
*/



#include <stddef.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include <unistd.h>
//#include "cuPrintf.cu"

//Your reads and refs should be files with size of 4 * 32 * 512 = 65536 lines
// /usr/local/cuda-10.1/bin/nvcc -D Nuints=8 -o Snake-on-GPU Snake-on-GPU.cu
//./Snake-on-GPU `head -n 1 ../Datasets/ERR240727_1_E2_30million.txt | awk '{print length($1)}'`  ../Datasets/ERR240727_1_E2_30million.txt  `wc -l  ../Datasets/ERR240727_1_E2_30million.txt|awk '{print $1}'`


#define warp_size 32
#define SharedPartDevice 64
#define FULL_MASK 0xffffffff
//#define NBytes 8
#define NBytes Nuints
#define PRINT 0

#define Number_of_Diagonals 9 //2*e+1
//#define F_ErrorThreshold 10
#define F_ReadLength 100

// #define NBytes 8

#define BitVal(data,y) ( (data>>y) & 1)      /** Return Data.Y value   **/
#define SetBit(data,y)    data |= (1 << y)    /** Set Data.Y   to 1    **/

// __device__ int popcount(int v) {
//     v = v - ((v >> 1) & 0x55555555);                // put count of each 2 bits into those 2 bits
//     v = (v & 0x33333333) + ((v >> 2) & 0x33333333); // put count of each 4 bits into those 4 bits
//     return ((v + (v >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
// }
// int popcount_Host(int v) {
//     v = v - ((v >> 1) & 0x55555555);                // put count of each 2 bits into those 2 bits
//     v = (v & 0x33333333) + ((v >> 2) & 0x33333333); // put count of each 4 bits into those 4 bits
//     return ((v + (v >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
// }

//__global__ void SneakySnake(uint* F_ReadSeq, uint* F_RefSeq, uint* Ftest_ReadSeq, uint* Ftest_RefSeq, int F_Number_of_Uints_inside_each_read, int F_Number_of_Reads_inside_each_cacheline, int F_Number_of_warps_inside_each_block,  int F_Number_of_blocks_inside_each_kernel, int* F_Results, int* Ftest_Results, int F_ErrorThreshold)
__global__ void SneakySnake(uint* F_ReadSeq, uint* F_RefSeq, int F_Number_of_Uints_inside_each_read, int F_Number_of_Reads_inside_each_cacheline, int F_Number_of_warps_inside_each_block,  int F_Number_of_blocks_inside_each_kernel, int* Ftest_Results, int F_ErrorThreshold, int NumReads2)
{

	// __shared__ uint SharedMemRefSeq[SharedPartDevice];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid>=NumReads2)
		return;
	//printf("%d",tid);

  // const int NBytes = 8;
  uint ReadsPerThread[NBytes];
  uint RefsPerThread[NBytes];

  #pragma unroll
  for (int i = 0; i < NBytes; i++)
  {
      ReadsPerThread[i] = F_ReadSeq[tid*8 + i];
      RefsPerThread[i] = F_RefSeq[tid*8 + i];

#if PRINT
			if(tid == 0)
			{
				printf("Read=%08x\t Ref=%08x\n", ReadsPerThread[i], RefsPerThread[i] );
			}
#endif

  }


////////////////////////////////////// Test: Reading from global memory to a reg, write back to global mem /////////////////////////////
  // uint readedReadSeq;
	// readedReadSeq = F_ReadSeq[tid];
	// Ftest_ReadSeq[tid] = readedReadSeq;
  //
  // uint readedRefSeq;
  // readedRefSeq = F_RefSeq[tid];
  // Ftest_RefSeq[tid] = readedRefSeq;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



  // if ((tid%1024) < SharedPartDevice)
	// {
	// 	SharedMemRefSeq[(tid%1024)] = F_RefSeq[(tid%1024)];
	// 	Ftest_RefSeq[(tid%1024)] = SharedMemRefSeq[(tid%1024)];
	// }

	// __syncthreads();


  /////////////////////////////////////////////////////////////////////////////
  Ftest_Results[tid] = 1;

  uint ReadCompTmp = 0x00000000;
  uint RefCompTmp = 0x00000000;
  uint DiagonalResult = 0x00000000;

  uint ReadTmp1 = 0x00000000;
  uint ReadTmp2 = 0x00000000;

  uint RefTmp1 = 0x00000000;
  uint RefTmp2 = 0x00000000;

  uint CornerCase = 0x00000000;

  //int localCounter[(2*F_ErrorThreshold)+1] = {0};
  int localCounter= 0;
  int localCounterMax=0;
  // int globalCounter_ref = 0; //just to find out which one of 8 uints we are using.
  int globalCounter = 0;

  int Max_leading_zeros = 0;

  // int Is_Edit = 0;
  // int Which_diag = 0;
  //int localErr = 0;
  int AccumulatedErrs = 0;

  // int ShiftValue_2Ref = 0;
  int ShiftValue = 0;
  // int ShiftCorrection = 0;

  int Diagonal = 0;

  // uint tmpEdits = 0x00000000;

  // int j_ref = 0;
  int j = 0; //specifying the j-th uint that we are reading in each read-ref comparison (can be from 0 to 7)

  // int TmpPrint = 0;

  while ( (j < 7) && (globalCounter < 200))
  {

#if PRINT
          if(tid == 0)
          {
            printf("#############\n");
          }
#endif

    Diagonal = 0;


    RefTmp1 = RefsPerThread[j] << ShiftValue; //SharedMemRefSeq[ ( ((tid%1024)%8) * 8) + j_ref] << ShiftValue_2Ref;
    RefTmp2 = RefsPerThread[j + 1] >>  32 - ShiftValue; // SharedMemRefSeq[ ( ((tid%1024)%8) * 8) + j_ref+1] >>  32 - ShiftValue_2Ref;

#if PRINT
          if(tid == 0)
          {
            printf("#Refs:\n");

            printf("RefNSh=%08x \t Sh=%d \t FTmp1=%08x\n", RefsPerThread[j], ShiftValue, RefTmp1);

            printf("RefNSh=%08x \t Sh=%d \t FTmp2=%08x\n\n", RefsPerThread[j + 1], 32 - ShiftValue, RefTmp2);
          }
#endif

    ReadTmp1 = ReadsPerThread[j] << ShiftValue;
    ReadTmp2 = ReadsPerThread[j + 1] >>  32 - ShiftValue;

#if PRINT
          if(tid == 0)
          {
            printf("#Main: j=%d\n", j);

            printf("NShJ=%08x \t Sh=%d \t RTmp1=%08x\n", ReadsPerThread[j], ShiftValue, ReadTmp1);

            printf("NShJ+1=%08x \t Sh=%d \t RTmp2=%08x\n", ReadsPerThread[j+1], 32 - ShiftValue, ReadTmp2);
          }
#endif

    ReadCompTmp = ReadTmp1 | ReadTmp2;
    RefCompTmp = RefTmp1 | RefTmp2;
    DiagonalResult = ReadCompTmp ^ RefCompTmp;
   // localCounter[Diagonal] = __clz(DiagonalResult);
	localCounterMax = __clz(DiagonalResult);

#if PRINT
			    if(tid == 0)
          {
            printf("RC=%08x \nFC=%08x \nDR=%08x \nD=%d \tLC[%d]=%d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounterMax);
          }
#endif

//////////////////// Upper diagonals /////////////////////

#if PRINT
          if(tid == 0)
          {
              printf("#upper\n");
          }
#endif

    for(int e = 1; e <= F_ErrorThreshold; e++)
    {
      Diagonal += 1;
      CornerCase = 0x00000000;
      if (  (j == 0)  &&  (  (ShiftValue - (2*e))  < 0 )  )
      {
        ReadTmp1 = ReadsPerThread[j] >> ( (2*e) - ShiftValue );
        ReadTmp2 = 0x00000000;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        CornerCase = 0x00000000;
        for(int Ci = 0; Ci < (2*e) - ShiftValue; Ci++)
        {
            SetBit(CornerCase, 31 - Ci);
        }

				DiagonalResult  = DiagonalResult | CornerCase;
        localCounter = __clz(DiagonalResult);

      }
      else if ( (ShiftValue - (2*e) ) < 0 )
      {
        ReadTmp1 = ReadsPerThread[j-1] << 32 - ( (2*e) - ShiftValue );
        ReadTmp2 = ReadsPerThread[j] >> (2*e) - ShiftValue;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        localCounter = __clz(DiagonalResult);

      }
      else
      {
        ReadTmp1 = ReadsPerThread[j] <<  ShiftValue - (2*e);
        ReadTmp2 = ReadsPerThread[j+1] >> 32 - (ShiftValue - (2*e) ) ;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        localCounter = __clz(DiagonalResult);

      }
		if (localCounter>localCounterMax)
			localCounterMax=localCounter;
#if PRINT
			if(tid == 0)
			{
					printf("j=%d, e=%d, sh=%d\n", j, e, ShiftValue - (2*e));
					printf("NShJ=%08x \t RTmp1=%08x\n", ReadsPerThread[j], ReadTmp1);
					printf("NShJ+1=%08x \t RTmp2=%08x\n", ReadsPerThread[j+1], ReadTmp2);
					printf("CC=%08x\n", CornerCase);
					printf("RC=%08x \nFC=%08x \nDR=%08x \nDN=%d \t LC[%d] = %d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounter);
					printf("\n");
			}
#endif

    }


/*
    sh = shift
    up = upper diagonal
    RC = ReadCompTmp
    FC = RefCompTmp
    D = DiagonalResult
    DN = diagonal
    LC = localCounter
*/

//////////////////// Lower diagonals /////////////////////

#if PRINT
		        if(tid == 0)
            {
                printf("#lower\n");
            }
#endif

    for(int e = 1; e <= F_ErrorThreshold; e++)
    {
		Diagonal += 1;
		CornerCase = 0x00000000;
		if ( j<5)//  ( (globalCounter + ShiftValue + (2*e) + 32) < 200) )
		{
			//printf("HI1");
			if ( (ShiftValue + (2*e) )  < 32)
			{
			  ReadTmp1 = ReadsPerThread[j] << ShiftValue + (2*e);
			  ReadTmp2 = ReadsPerThread[j+1] >> 32 - ( ShiftValue + (2*e) );
						// ReadTmp2 = 0x00000000;

						ReadCompTmp = ReadTmp1 | ReadTmp2;
						RefCompTmp = RefTmp1 | RefTmp2;

						DiagonalResult = ReadCompTmp ^ RefCompTmp;
			  localCounter = __clz(DiagonalResult);

			}
			else
			{
				//printf("HI2");
			  ReadTmp1 = ReadsPerThread[j+1] << ( ShiftValue + (2*e) ) % 32;
			  ReadTmp2 = ReadsPerThread[j+2] >>  32 - ( ( ShiftValue + (2*e) ) % 32 );

			  ReadCompTmp = ReadTmp1 | ReadTmp2;
			  RefCompTmp = RefTmp1 | RefTmp2;

			  DiagonalResult = 0xffffffff;//ReadCompTmp ^ RefCompTmp;

			  DiagonalResult = ReadCompTmp ^ RefCompTmp;

			  localCounter = __clz(DiagonalResult);
			}
		}
		else
		{
			//printf("HI3");
			ReadTmp1 = ReadsPerThread[j] << ShiftValue + (2*e);
			ReadTmp2 = ReadsPerThread[j+1] >>   32 - ( ShiftValue + (2*e) );

			ReadCompTmp = ReadTmp1 | ReadTmp2;
			RefCompTmp = RefTmp1 | RefTmp2;
			DiagonalResult = ReadCompTmp ^ RefCompTmp;

			CornerCase = 0x00000000;
			if ((globalCounter+32)>200 ) {
			
				for(int Ci = ((globalCounter+32)-200); Ci < (((globalCounter+32)-200)+ 2*e); Ci++)
				{
				  SetBit(CornerCase, Ci);
				}
			}

			else if ((globalCounter+32)>=(200- (2*e))){
			
				for(int Ci = 0; Ci < (2*e); Ci++)
				{
				  SetBit(CornerCase, Ci);
				}
			}
			DiagonalResult = DiagonalResult | CornerCase;
			
			localCounter = __clz(DiagonalResult);
      }
	  
	  if (localCounter>localCounterMax)
			localCounterMax=localCounter;

#if PRINT
			if(tid == 0)
			{
				// printf("aaaaaaaaaa\n");
				printf("j=%d, e=%d, Sh=%d\n", j, e, ShiftValue + (2*e));
				printf("NShJ=%08x \t RTmp1=%08x\n", ReadsPerThread[j], ReadTmp1);
				printf("NShJ+1=%08x \t RTmp2=%08x\n", ReadsPerThread[j+1], ReadTmp2);
				printf("CC=%08x\n", CornerCase);
				printf("RC=%08x \nFC=%08x \nDR=%08x \nDN=%d \t LC[%d] = %d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounter);
				printf("\n");
			}
#endif

    }

    /*
    CC = CornerCase
        sh = shift
        up = upper diagonal
        RC = ReadCompTmp
        FC = RefCompTmp
        D = DiagonalResult
        DN = diagonal
        LC = localCounter
    */

    Max_leading_zeros = 0;
    // Is_Edit = 0;
    // Which_diag = 0;


	if ( (j == 6) && ( ((localCounterMax/2)*2) >= 8)  )
	{
		Max_leading_zeros = 8;
		// if ( (Max_leading_zeros != 8) && (tmp > 0) )
		// {
		//     Is_Edit = 1;
		//     Which_diag = tmp;
		// }
		break;
	}
	else if( ((localCounterMax/2)*2) > Max_leading_zeros)
	{
		Max_leading_zeros = ((localCounterMax/2)*2);
		// if (tmp > 0)
		// {
		//     Is_Edit = 1;
		//     Which_diag = tmp;
		// }
	}
    

    // int Considered_Edits = 0;
		//
    // if ( (Is_Edit == 1) && (Which_diag <= (Number_of_Diagonals/2) ) )
    // {
    //   Considered_Edits = Which_diag;
    // }
    // else if(Is_Edit == 1)
    // {
    //   Considered_Edits = Which_diag - (Number_of_Diagonals/2);
    // }


    if ( ( (Max_leading_zeros/2) < 16) && (j < 5) )
    {
      AccumulatedErrs += 1;
    }
    else if (  (j == 6) && ( (Max_leading_zeros/2) < 4) )
    {
      AccumulatedErrs += 1;
    }
    // else if ( ( (Max_leading_zeros/2) == 16) && (Is_Edit == 1) && (j < 5) )
    // {
    //   AccumulatedErrs += 1;
    // }

#if PRINT
            if(tid == 0)
            {
              printf("Diag finished\n");
							printf("Max=%d \n AccErr=%d\n", Max_leading_zeros, AccumulatedErrs);
              // printf("Is_Edit=%d \t Wh=%d \t Max=%d \n AccErr=%d\n", Is_Edit, Which_diag, Max_leading_zeros, AccumulatedErrs);
            }
#endif

    if(AccumulatedErrs > F_ErrorThreshold)
    {
      Ftest_Results[tid] = 0;
      break;
    }


    if(ShiftValue + Max_leading_zeros + 2 >= 32)
    {
      j += 1;
      // j_ref += 1;
    }

    // ShiftValue_2Ref = (ShiftValue_2Ref + Max_leading_zeros + 2) %32;
    if (Max_leading_zeros == 32)
    {
			globalCounter += Max_leading_zeros;
    }
    else
    {
        ShiftValue = ((ShiftValue + Max_leading_zeros + 2) % 32);
				globalCounter += (Max_leading_zeros + 2);
    }



#if PRINT
            if(tid == 0)
            {
              printf("GC=%d\n", globalCounter);
              printf("Expected shift = %d\n\n\n", ShiftValue);
            }
#endif

  }
// __syncthreads();
}
