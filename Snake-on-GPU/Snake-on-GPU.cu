#include "hip/hip_runtime.h"
/*
 * Copyright (c) <2017 - 2030>, ETH Zurich and Bilkent University
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * Redistributions of source code must retain the above copyright notice, this list
 * of conditions and the following disclaimer.
 * - Redistributions in binary form must reproduce the above copyright notice, this
 *   list of conditions and the following disclaimer in the documentation and/or other
 *   materials provided with the distribution.
 * - Neither the names of the ETH Zurich, Bilkent University,
 *   nor the names of its contributors may be
 *   used to endorse or promote products derived from this software without specific
 *   prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 * LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  Authors: 
  Mohammed Alser
	  mohammed.alser AT inf DOT ethz DOT ch
  Date:
  September 22nd, 2019
*/

#include <stddef.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include <unistd.h>
//#include "cuPrintf.cu"

//Your reads and refs should be files with size of 4 * 32 * 512 = 65536 lines
// /usr/local/cuda-10.1/bin/nvcc -D Nuints=8 -o Snake-on-GPU Snake-on-GPU.cu
//./Snake-on-GPU `head -n 1 ../Datasets/ERR240727_1_E2_30million.txt | awk '{print length($1)}'`  ../Datasets/ERR240727_1_E2_30million.txt  `wc -l  ../Datasets/ERR240727_1_E2_30million.txt|awk '{print $1}'`


#define warp_size 32
#define SharedPartDevice 64
#define FULL_MASK 0xffffffff
//#define NBytes 8
#define NBytes Nuints
#define PRINT 0

#define Number_of_Diagonals 9 //2*e+1
//#define F_ErrorThreshold 10
#define F_ReadLength 100

// #define NBytes 8

#define BitVal(data,y) ( (data>>y) & 1)      /** Return Data.Y value   **/
#define SetBit(data,y)    data |= (1 << y)    /** Set Data.Y   to 1    **/

// __device__ int popcount(int v) {
//     v = v - ((v >> 1) & 0x55555555);                // put count of each 2 bits into those 2 bits
//     v = (v & 0x33333333) + ((v >> 2) & 0x33333333); // put count of each 4 bits into those 4 bits
//     return ((v + (v >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
// }
// int popcount_Host(int v) {
//     v = v - ((v >> 1) & 0x55555555);                // put count of each 2 bits into those 2 bits
//     v = (v & 0x33333333) + ((v >> 2) & 0x33333333); // put count of each 4 bits into those 4 bits
//     return ((v + (v >> 4) & 0xF0F0F0F) * 0x1010101) >> 24;
// }

//__global__ void SneakySnake(uint* F_ReadSeq, uint* F_RefSeq, uint* Ftest_ReadSeq, uint* Ftest_RefSeq, int F_Number_of_Uints_inside_each_read, int F_Number_of_Reads_inside_each_cacheline, int F_Number_of_warps_inside_each_block,  int F_Number_of_blocks_inside_each_kernel, int* F_Results, int* Ftest_Results, int F_ErrorThreshold)
__global__ void SneakySnake(uint* F_ReadSeq, uint* F_RefSeq, int F_Number_of_Uints_inside_each_read, int F_Number_of_Reads_inside_each_cacheline, int F_Number_of_warps_inside_each_block,  int F_Number_of_blocks_inside_each_kernel, int* Ftest_Results, int F_ErrorThreshold, int NumReads2)
{

	// __shared__ uint SharedMemRefSeq[SharedPartDevice];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid>=NumReads2)
		return;
	//printf("%d",tid);

  // const int NBytes = 8;
  uint ReadsPerThread[NBytes];
  uint RefsPerThread[NBytes];

  #pragma unroll
  for (int i = 0; i < NBytes; i++)
  {
      ReadsPerThread[i] = F_ReadSeq[tid*8 + i];
      RefsPerThread[i] = F_RefSeq[tid*8 + i];

#if PRINT
			if(tid == 0)
			{
				printf("Read=%08x\t Ref=%08x\n", ReadsPerThread[i], RefsPerThread[i] );
			}
#endif

  }


////////////////////////////////////// Test: Reading from global memory to a reg, write back to global mem /////////////////////////////
  // uint readedReadSeq;
	// readedReadSeq = F_ReadSeq[tid];
	// Ftest_ReadSeq[tid] = readedReadSeq;
  //
  // uint readedRefSeq;
  // readedRefSeq = F_RefSeq[tid];
  // Ftest_RefSeq[tid] = readedRefSeq;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



  // if ((tid%1024) < SharedPartDevice)
	// {
	// 	SharedMemRefSeq[(tid%1024)] = F_RefSeq[(tid%1024)];
	// 	Ftest_RefSeq[(tid%1024)] = SharedMemRefSeq[(tid%1024)];
	// }

	// __syncthreads();


  /////////////////////////////////////////////////////////////////////////////
  Ftest_Results[tid] = 1;

  uint ReadCompTmp = 0x00000000;
  uint RefCompTmp = 0x00000000;
  uint DiagonalResult = 0x00000000;

  uint ReadTmp1 = 0x00000000;
  uint ReadTmp2 = 0x00000000;

  uint RefTmp1 = 0x00000000;
  uint RefTmp2 = 0x00000000;

  uint CornerCase = 0x00000000;

  //int localCounter[(2*F_ErrorThreshold)+1] = {0};
  int localCounter= 0;
  int localCounterMax=0;
  // int globalCounter_ref = 0; //just to find out which one of 8 uints we are using.
  int globalCounter = 0;

  int Max_leading_zeros = 0;

  // int Is_Edit = 0;
  // int Which_diag = 0;
  //int localErr = 0;
  int AccumulatedErrs = 0;

  // int ShiftValue_2Ref = 0;
  int ShiftValue = 0;
  // int ShiftCorrection = 0;

  int Diagonal = 0;

  // uint tmpEdits = 0x00000000;

  // int j_ref = 0;
  int j = 0; //specifying the j-th uint that we are reading in each read-ref comparison (can be from 0 to 7)

  // int TmpPrint = 0;

  while ( (j < 7) && (globalCounter < 200))
  {

#if PRINT
          if(tid == 0)
          {
            printf("#############\n");
          }
#endif

    Diagonal = 0;


    RefTmp1 = RefsPerThread[j] << ShiftValue; //SharedMemRefSeq[ ( ((tid%1024)%8) * 8) + j_ref] << ShiftValue_2Ref;
    RefTmp2 = RefsPerThread[j + 1] >>  32 - ShiftValue; // SharedMemRefSeq[ ( ((tid%1024)%8) * 8) + j_ref+1] >>  32 - ShiftValue_2Ref;

#if PRINT
          if(tid == 0)
          {
            printf("#Refs:\n");

            printf("RefNSh=%08x \t Sh=%d \t FTmp1=%08x\n", RefsPerThread[j], ShiftValue, RefTmp1);

            printf("RefNSh=%08x \t Sh=%d \t FTmp2=%08x\n\n", RefsPerThread[j + 1], 32 - ShiftValue, RefTmp2);
          }
#endif

    ReadTmp1 = ReadsPerThread[j] << ShiftValue;
    ReadTmp2 = ReadsPerThread[j + 1] >>  32 - ShiftValue;

#if PRINT
          if(tid == 0)
          {
            printf("#Main: j=%d\n", j);

            printf("NShJ=%08x \t Sh=%d \t RTmp1=%08x\n", ReadsPerThread[j], ShiftValue, ReadTmp1);

            printf("NShJ+1=%08x \t Sh=%d \t RTmp2=%08x\n", ReadsPerThread[j+1], 32 - ShiftValue, ReadTmp2);
          }
#endif

    ReadCompTmp = ReadTmp1 | ReadTmp2;
    RefCompTmp = RefTmp1 | RefTmp2;
    DiagonalResult = ReadCompTmp ^ RefCompTmp;
   // localCounter[Diagonal] = __clz(DiagonalResult);
	localCounterMax = __clz(DiagonalResult);

#if PRINT
			    if(tid == 0)
          {
            printf("RC=%08x \nFC=%08x \nDR=%08x \nD=%d \tLC[%d]=%d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounterMax);
          }
#endif

//////////////////// Upper diagonals /////////////////////

#if PRINT
          if(tid == 0)
          {
              printf("#upper\n");
          }
#endif

    for(int e = 1; e <= F_ErrorThreshold; e++)
    {
      Diagonal += 1;
      CornerCase = 0x00000000;
      if (  (j == 0)  &&  (  (ShiftValue - (2*e))  < 0 )  )
      {
        ReadTmp1 = ReadsPerThread[j] >> ( (2*e) - ShiftValue );
        ReadTmp2 = 0x00000000;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        CornerCase = 0x00000000;
        for(int Ci = 0; Ci < (2*e) - ShiftValue; Ci++)
        {
            SetBit(CornerCase, 31 - Ci);
        }

				DiagonalResult  = DiagonalResult | CornerCase;
        localCounter = __clz(DiagonalResult);

      }
      else if ( (ShiftValue - (2*e) ) < 0 )
      {
        ReadTmp1 = ReadsPerThread[j-1] << 32 - ( (2*e) - ShiftValue );
        ReadTmp2 = ReadsPerThread[j] >> (2*e) - ShiftValue;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        localCounter = __clz(DiagonalResult);

      }
      else
      {
        ReadTmp1 = ReadsPerThread[j] <<  ShiftValue - (2*e);
        ReadTmp2 = ReadsPerThread[j+1] >> 32 - (ShiftValue - (2*e) ) ;

        ReadCompTmp = ReadTmp1 | ReadTmp2;
        RefCompTmp = RefTmp1 | RefTmp2;

        DiagonalResult = ReadCompTmp ^ RefCompTmp;

        localCounter = __clz(DiagonalResult);

      }
		if (localCounter>localCounterMax)
			localCounterMax=localCounter;
#if PRINT
			if(tid == 0)
			{
					printf("j=%d, e=%d, sh=%d\n", j, e, ShiftValue - (2*e));
					printf("NShJ=%08x \t RTmp1=%08x\n", ReadsPerThread[j], ReadTmp1);
					printf("NShJ+1=%08x \t RTmp2=%08x\n", ReadsPerThread[j+1], ReadTmp2);
					printf("CC=%08x\n", CornerCase);
					printf("RC=%08x \nFC=%08x \nDR=%08x \nDN=%d \t LC[%d] = %d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounter);
					printf("\n");
			}
#endif

    }


/*
    sh = shift
    up = upper diagonal
    RC = ReadCompTmp
    FC = RefCompTmp
    D = DiagonalResult
    DN = diagonal
    LC = localCounter
*/

//////////////////// Lower diagonals /////////////////////

#if PRINT
		        if(tid == 0)
            {
                printf("#lower\n");
            }
#endif

    for(int e = 1; e <= F_ErrorThreshold; e++)
    {
		Diagonal += 1;
		CornerCase = 0x00000000;
		if ( j<5)//  ( (globalCounter + ShiftValue + (2*e) + 32) < 200) )
		{
			//printf("HI1");
			if ( (ShiftValue + (2*e) )  < 32)
			{
			  ReadTmp1 = ReadsPerThread[j] << ShiftValue + (2*e);
			  ReadTmp2 = ReadsPerThread[j+1] >> 32 - ( ShiftValue + (2*e) );
						// ReadTmp2 = 0x00000000;

						ReadCompTmp = ReadTmp1 | ReadTmp2;
						RefCompTmp = RefTmp1 | RefTmp2;

						DiagonalResult = ReadCompTmp ^ RefCompTmp;
			  localCounter = __clz(DiagonalResult);

			}
			else
			{
				//printf("HI2");
			  ReadTmp1 = ReadsPerThread[j+1] << ( ShiftValue + (2*e) ) % 32;
			  ReadTmp2 = ReadsPerThread[j+2] >>  32 - ( ( ShiftValue + (2*e) ) % 32 );

			  ReadCompTmp = ReadTmp1 | ReadTmp2;
			  RefCompTmp = RefTmp1 | RefTmp2;

			  DiagonalResult = 0xffffffff;//ReadCompTmp ^ RefCompTmp;

			  DiagonalResult = ReadCompTmp ^ RefCompTmp;

			  localCounter = __clz(DiagonalResult);
			}
		}
		else
		{
			//printf("HI3");
			ReadTmp1 = ReadsPerThread[j] << ShiftValue + (2*e);
			ReadTmp2 = ReadsPerThread[j+1] >>   32 - ( ShiftValue + (2*e) );

			ReadCompTmp = ReadTmp1 | ReadTmp2;
			RefCompTmp = RefTmp1 | RefTmp2;
			DiagonalResult = ReadCompTmp ^ RefCompTmp;

			CornerCase = 0x00000000;
			if ((globalCounter+32)>200 ) {
			
				for(int Ci = ((globalCounter+32)-200); Ci < (((globalCounter+32)-200)+ 2*e); Ci++)
				{
				  SetBit(CornerCase, Ci);
				}
			}

			else if ((globalCounter+32)>=(200- (2*e))){
			
				for(int Ci = 0; Ci < (2*e); Ci++)
				{
				  SetBit(CornerCase, Ci);
				}
			}
			DiagonalResult = DiagonalResult | CornerCase;
			
			localCounter = __clz(DiagonalResult);
      }
	  
	  if (localCounter>localCounterMax)
			localCounterMax=localCounter;

#if PRINT
			if(tid == 0)
			{
				// printf("aaaaaaaaaa\n");
				printf("j=%d, e=%d, Sh=%d\n", j, e, ShiftValue + (2*e));
				printf("NShJ=%08x \t RTmp1=%08x\n", ReadsPerThread[j], ReadTmp1);
				printf("NShJ+1=%08x \t RTmp2=%08x\n", ReadsPerThread[j+1], ReadTmp2);
				printf("CC=%08x\n", CornerCase);
				printf("RC=%08x \nFC=%08x \nDR=%08x \nDN=%d \t LC[%d] = %d\n\n", ReadCompTmp, RefCompTmp, DiagonalResult, Diagonal, Diagonal, localCounter);
				printf("\n");
			}
#endif

    }

    /*
    CC = CornerCase
        sh = shift
        up = upper diagonal
        RC = ReadCompTmp
        FC = RefCompTmp
        D = DiagonalResult
        DN = diagonal
        LC = localCounter
    */

    Max_leading_zeros = 0;
    // Is_Edit = 0;
    // Which_diag = 0;


	if ( (j == 6) && ( ((localCounterMax/2)*2) >= 8)  )
	{
		Max_leading_zeros = 8;
		// if ( (Max_leading_zeros != 8) && (tmp > 0) )
		// {
		//     Is_Edit = 1;
		//     Which_diag = tmp;
		// }
		break;
	}
	else if( ((localCounterMax/2)*2) > Max_leading_zeros)
	{
		Max_leading_zeros = ((localCounterMax/2)*2);
		// if (tmp > 0)
		// {
		//     Is_Edit = 1;
		//     Which_diag = tmp;
		// }
	}
    

    // int Considered_Edits = 0;
		//
    // if ( (Is_Edit == 1) && (Which_diag <= (Number_of_Diagonals/2) ) )
    // {
    //   Considered_Edits = Which_diag;
    // }
    // else if(Is_Edit == 1)
    // {
    //   Considered_Edits = Which_diag - (Number_of_Diagonals/2);
    // }


    if ( ( (Max_leading_zeros/2) < 16) && (j < 5) )
    {
      AccumulatedErrs += 1;
    }
    else if (  (j == 6) && ( (Max_leading_zeros/2) < 4) )
    {
      AccumulatedErrs += 1;
    }
    // else if ( ( (Max_leading_zeros/2) == 16) && (Is_Edit == 1) && (j < 5) )
    // {
    //   AccumulatedErrs += 1;
    // }

#if PRINT
            if(tid == 0)
            {
              printf("Diag finished\n");
							printf("Max=%d \n AccErr=%d\n", Max_leading_zeros, AccumulatedErrs);
              // printf("Is_Edit=%d \t Wh=%d \t Max=%d \n AccErr=%d\n", Is_Edit, Which_diag, Max_leading_zeros, AccumulatedErrs);
            }
#endif

    if(AccumulatedErrs > F_ErrorThreshold)
    {
      Ftest_Results[tid] = 0;
      break;
    }


    if(ShiftValue + Max_leading_zeros + 2 >= 32)
    {
      j += 1;
      // j_ref += 1;
    }

    // ShiftValue_2Ref = (ShiftValue_2Ref + Max_leading_zeros + 2) %32;
    if (Max_leading_zeros == 32)
    {
			globalCounter += Max_leading_zeros;
    }
    else
    {
        ShiftValue = ((ShiftValue + Max_leading_zeros + 2) % 32);
				globalCounter += (Max_leading_zeros + 2);
    }



#if PRINT
            if(tid == 0)
            {
              printf("GC=%d\n", globalCounter);
              printf("Expected shift = %d\n\n\n", ShiftValue);
            }
#endif

  }
// __syncthreads();
}

void cudaCheckError(hipError_t cudaStatus, const char* err)
{
    if(cudaStatus != hipSuccess)
    {
        fprintf(stderr, err);
        hipDeviceReset();
       exit(EXIT_FAILURE);
    }
}


int main(int argc, const char * const argv[])
{
	if (argc != 4){
		printf("Incorrect arguments..\n./Snake-on-GPU [ReadLength] [ReadandRefFile] [#reads]\n");
		exit(-1);
	}

	// int DebugMode=atoi(argv[1]);//For me is always 1; Never debugging :-D
	// int GridSize=atoi(argv[2]);//I basically never used this variable. Just left it for consistance of inputs with the c code.
	// int KmerSize=atoi(argv[3]);//This is the size of the sliding window (usually a multiples of ReadLength)
	int ReadLength = atoi(argv[1]);//in my inputs, it is always 100. Just for the generality we keep it as a variable
	int NumReads = atoi(argv[3]); // Number of reads
	//int F_ErrorThreshold = atoi(argv[2]);
	// int NumberOfFilesLines = atoi(argv[7]);//This variable shows the total number of readed lines from the input
	// int IterationNo= atoi(argv[8]);//Mohammed iterates at least IterationNo times inside his function. Not sure why!
	// int Concurrent_Lines_Reading = atoi(argv[9]);//This was originally used to show the number of concurrent lines

	// int Cache_line_size = 1024; //in bits
	// int Size_of_each_read = 256; //in bits
	int Size_of_uint_in_Bit = 32; //in Bits 
	// int Size_of_uint_in_Byte = (int)sizeof(uint);

	//uint print = 0x00000001;
	//printf ("%08x\n ", print);

	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	char *p;//when reading each char_basepair from the file, we read it into the p.

	int Number_of_Uints_inside_each_read = 8;
	int Number_of_Reads_inside_each_cacheline = 4;
	int Number_of_warps_inside_each_block = 32; //assuming that the block size is 1024 we have 32 warp inside each block! (Now is 1 for testing)
	//int Number_of_blocks_inside_each_kernel = 512;
	int Concurrent_threads_In_Block = warp_size * Number_of_warps_inside_each_block;
	int Number_of_blocks_inside_each_kernel = ceil(NumReads / Concurrent_threads_In_Block);

	//int Concurrent_threads_In_Block = Number_of_Reads_inside_each_cacheline * Number_of_warps_inside_each_block;
	int F_ErrorThreshold =0;
	// int Read_number_inside_each_cacheline = 0; //from 0 to 4
	// int Uint_number_inside_each_read = 0; //from 0 to 8
	// int Bit_number_inside_each_uint = 0; //from 0 to 32

	//printf("Cache_line_size = %d\nSize_of_each_read = %d\nSize_of_uint_in_Bit = %d\nSize_of_uint_in_Byte = %d\nNumber_of_Reads_inside_each_cacheline = %d\nNumber_of_Uints_inside_each_read = %d\nNumber_of_warps_inside_each_block = %d\nRead_number_inside_each_cacheline = %d\nUint_number_inside_each_read = %d\nBit_number_inside_each_uint = %d\n", Cache_line_size, Size_of_each_read, Size_of_uint_in_Bit, Size_of_uint_in_Byte, Number_of_Reads_inside_each_cacheline, Number_of_Uints_inside_each_read, Number_of_warps_inside_each_block, Read_number_inside_each_cacheline, Uint_number_inside_each_read, Bit_number_inside_each_uint);
	int loopPar = 0;
	for (loopPar =0; loopPar<=10;loopPar++) {
		F_ErrorThreshold=(loopPar*ReadLength)/100;

		int devID;
		hipDeviceProp_t deviceProps;

		int device_count;
		cudaCheckError(hipGetDeviceCount(&device_count), "Couldn't get device count!");

		if (device_count == 0)
		{
			fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
			exit(EXIT_FAILURE);
		}

		devID = 0;
		cudaCheckError(hipSetDevice(devID), "Couldn't set device!");
		cudaCheckError(hipGetDeviceProperties(&deviceProps, devID), "Couldn't get Device Properties");
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProps.name, deviceProps.major, deviceProps.minor);

		//hipDeviceReset();
		///////////required arrays inside the host
		uint* ReadSeq;
		//ReadSeq = (uint * ) calloc(((NumReads * ReadLength * 2) + 8*sizeof(uint)) / 8*sizeof(uint), sizeof(uint));
		ReadSeq = (uint * ) calloc(NumReads * 8, sizeof(uint));
		uint* RefSeq;
		RefSeq = (uint * ) calloc(NumReads * 8, sizeof(uint));
		///////////////////////////////////////////////////  Our memory on the GPU side //////////////////////////////////////////////////////////////
		//////////Required arrays in the device
		uint* Dev_ReadSeq;
		//Dev_ReadSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));
		uint* Dev_RefSeq;
		//Dev_RefSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));
		int* Dev_Results;
		//Dev_Results = (int *) calloc(resultSize, sizeof(int));
		/////////Mallocing on the device side
		//hipMalloc((uint**)&Dev_ReadSeq, sizeof(uint)*UintsReadsWarpsBlocks);
		//hipMalloc((uint**)&Dev_RefSeq, sizeof(uint)*UintsReadsWarpsBlocks);
		//hipMalloc((int**)&Dev_Results, sizeof(int)*resultSize);
		hipMalloc((uint**)&Dev_ReadSeq, (NumReads * 8 * sizeof(uint)));
		hipMalloc((uint**)&Dev_RefSeq, (NumReads * 8 * sizeof(uint)));
		hipMalloc((int**)&Dev_Results, NumReads * sizeof(int));

		////////////////////////////////////////////////// test purposes     ///////////////////////////////////////////////////////////////
		/*uint* Dtest_ReadSeq;
		Dtest_ReadSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));
		uint* Dtest_RefSeq;
		Dtest_RefSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));
		hipMalloc((uint**)&Dtest_ReadSeq, sizeof(uint)*UintsReadsWarpsBlocks);
		hipMalloc((uint**)&Dtest_RefSeq, sizeof(uint)*UintsReadsWarpsBlocks);*/

		int* Dtest_Results;
		//Dtest_Results = (int *) calloc(resultSize, sizeof(int));
		hipMalloc((int**)&Dtest_Results, NumReads * sizeof(int));

		////////////////////////////////////////////////// Final results    ///////////////////////////////////////////////////////////////
		/*uint* DFinal_ReadSeq;
		DFinal_ReadSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));
		uint* DFinal_RefSeq;
		DFinal_RefSeq = (uint * ) calloc(UintsReadsWarpsBlocks, sizeof(uint));*/

		int* DFinal_Results;
		//DFinal_Results = (int * ) calloc(resultSize, sizeof(int));
		DFinal_Results = (int * ) calloc(NumReads, sizeof(int));
		/////////////////////////////////////////////////////////////////////////////////////////////////////////////////

		//printf("aaaa = %d\n", ((20%(Size_of_uint_in_Bit/2)) * 2));

		// uint test = 0x00000000;
		// test = SetBit(test, 31 - 6);
		// printf("%08x\n", test);
		int tokenIndex=1;
		fp = fopen(argv[2], "r");
		if (!fp){
			printf("Sorry, the file does not exist or you do not have access permission\n");
			return 0;
		}
		for(int this_read = 0; this_read < NumReads; this_read++){
			read = getline(&line, &len, fp);
			tokenIndex=1;
			for (p = strtok(line, "\t"); p != NULL; p = strtok(NULL, "\t"))
			{
				if (tokenIndex==1)
				{
					for (int j = 0; j < ReadLength; j++)
					{
						if(p[j] == 'A')
						{
							//do nothing (this is like storing 00)
						}
						else if (p[j] == 'C')
						{
							ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2 + 1));
						}
						else if (p[j] == 'G')
						{
							ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2));
						}
						else if (p[j] == 'T')
						{
							ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2));

							ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2 + 1));
						}
						//printf("%c",p[j]);
						//printf(" %08x", ReadSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)]);
					}
				}
				else if(tokenIndex==2)
				{
					for (int j = 0; j < ReadLength; j++)
					{
						if(p[j] == 'A')
						{
							//do nothing (this is like storing 00)
						}
						else if (p[j] == 'C')
						{
							RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2 + 1));
						}
						else if (p[j] == 'G')
						{
							RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2));
						}
						else if (p[j] == 'T')
						{
							RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2));

							RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)] = SetBit(RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)], 31 - ((j%(Size_of_uint_in_Bit/2)) * 2 + 1));
						}
						//printf("%c",p[j]);
						//printf(" %08x", RefSeq[((j*2/Size_of_uint_in_Bit) + this_read * NBytes)]);
					}
				}
				tokenIndex=tokenIndex+1;
			//printf("\n");
			}
		}
	/*	for(int this_read = 0; this_read < 15; this_read++){
		printf(" %08x", ReadSeq[this_read]);
		}*/
		fclose(fp);

		

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);


		hipEventRecord(start);
		
		//hipMemcpy(Dev_ReadSeq, ReadSeq, sizeof(uint)*UintsReadsWarpsBlocks, hipMemcpyHostToDevice);
		//hipMemcpy(Dev_RefSeq, RefSeq, sizeof(uint)*UintsReadsWarpsBlocks, hipMemcpyHostToDevice);
		//hipMemcpy(Dev_Results, Results, sizeof(int)*resultSize, hipMemcpyHostToDevice);
		hipMemcpy(Dev_ReadSeq, ReadSeq, NumReads * 8 * sizeof(uint), hipMemcpyHostToDevice);
		hipMemcpy(Dev_RefSeq, RefSeq, NumReads * 8 * sizeof(uint), hipMemcpyHostToDevice);
		//hipMemcpy(Dev_Results, Results, sizeof(int)*NumReads, hipMemcpyHostToDevice);

		// int accepted = 0;
		// for(int i = 0; i < resultSize; i++)
		// {
		//   if(DFinal_Results[i] == 1)
		//     accepted += 1;
		//   // printf("%d \t %d\n", i+1, DFinal_Results[i]);
		// }
		//
		// printf("E: \t 4 \t Accepted: \t %10d \t Rejected: \t %10d\n", accepted, resultSize - accepted);

		
		//SneakySnake<<<Number_of_blocks_inside_each_kernel,Concurrent_threads_In_Block>>>(Dev_ReadSeq, Dev_RefSeq, Dtest_ReadSeq, Dtest_RefSeq, Number_of_Uints_inside_each_read, Number_of_Reads_inside_each_cacheline, Number_of_warps_inside_each_block,  Number_of_blocks_inside_each_kernel, Dev_Results, Dtest_Results, F_ErrorThreshold);
		SneakySnake<<<Number_of_blocks_inside_each_kernel,Concurrent_threads_In_Block>>>(Dev_ReadSeq, Dev_RefSeq, Number_of_Uints_inside_each_read, Number_of_Reads_inside_each_cacheline, Number_of_warps_inside_each_block,  Number_of_blocks_inside_each_kernel, Dtest_Results, F_ErrorThreshold, NumReads);

		hipDeviceSynchronize();

		// hipMemcpy(DFinal_ReadSeq, Dtest_ReadSeq, sizeof(uint)*UintsReadsWarpsBlocks, hipMemcpyDeviceToHost);
		// hipMemcpy(DFinal_RefSeq, Dtest_RefSeq, sizeof(uint)*UintsReadsWarpsBlocks, hipMemcpyDeviceToHost);
		hipMemcpy(DFinal_Results, Dtest_Results, sizeof(int)*NumReads, hipMemcpyDeviceToHost);

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		int accepted = 0;
		for(int i = 0; i < NumReads; i++)
		{
		if(DFinal_Results[i] == 1)
		  accepted += 1;
		 //printf("%d \t %d\n", i+1, DFinal_Results[i]);
		//printf("%d\n", DFinal_Results[i]);
		}

		printf("E: \t %d \t Snake-on-GPU: \t %5.4f \t Accepted: \t %10d \t Rejected: \t %10d\n", F_ErrorThreshold, milliseconds, accepted, NumReads - accepted);


		// printf("#############    We are printing readed back (Copied inside GPU) ReadSeq in CPU side  ###############\n");
		// for(int i = 0; i < 8; i++)
		// {
		// 		printf("%08x\n", DFinal_ReadSeq[i]);
		// }
		// printf("##########   We are printing readed back (Copied inside GPU) RefSeq in CPU side #######################\n");
		// for(int i = 0; i < 8; i++)
		// {
		// 	printf("%08x\n", DFinal_RefSeq[i]);
		// }
	}
	return 0;
}
